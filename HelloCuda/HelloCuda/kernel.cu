#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

using namespace std;

__device__ int add_one(int a) {

	return a + 1;
}

__global__ void show(int *a) {

	for (int i = 0; i < 10; i++) {

		//a[i] = add_one(a[i]);
		printf(" %d", a[i]);
	}
	printf("\n");
}

__global__ void changeValue(int *a) {

	for (int i = 0; i < 10; i++) {

		a[i] = 100;
	}
}

int main() {

	int cpu_int[10] = {10, 10, 10, 10, 10, 10, 10, 10, 10, 10};
	int *gpu_int;

	//��GPU�Ϸ���ռ�
	hipMalloc((void**)&gpu_int, 10 * sizeof(int));
	show << <1, 1 >> > (gpu_int);

	//��ʼ����ֵ
	hipMemset(gpu_int, 0, 10 * sizeof(int));
	show<< <1, 1 >> > (gpu_int);

	//��cpu_int��ֵ��gpu_int
	hipMemcpy(gpu_int, cpu_int, 10 * sizeof(int), hipMemcpyHostToDevice);
	show << <1, 1 >> > (gpu_int);

	//�ı�gpu_int��ֵ
	changeValue << <1, 1 >> >(gpu_int);
	show << <1, 1 >> > (gpu_int);


	hipMemcpy(cpu_int, gpu_int, 10 * sizeof(int), hipMemcpyDeviceToHost);
	printf("----------�����ķָ���----------");
	for (int i = 0; i < 10; i++) {


		printf("%d", cpu_int[i]);
	}



	hipFree(gpu_int);


	getchar();
	return 0;
}